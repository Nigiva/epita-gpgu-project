#include "hip/hip_runtime.h"
#include "render.hpp"
#include <spdlog/spdlog.h>
#include <cassert>
#include "utils.hpp"

    [[gnu::noinline]]
void _abortError(const char* msg, const char* fname, int line)
{
    hipError_t err = hipGetLastError();
    spdlog::error("{} ({}, line: {})", msg, fname, line);
    spdlog::error("Error {}: {}", hipGetErrorName(err), hipGetErrorString(err));
    std::exit(1);
}

#define abortError(msg) _abortError(msg, __FUNCTION__, __LINE__)

// Device code
__global__ void gpu_gaussian_blur(char* img_buffer, int width, int height, size_t img_pitch, double* gaussian_kernel, int kernel_size, char* res_buffer, size_t res_pitch)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    int mid_kernel = (kernel_size - 1) / 2;

    // get pixel value
    char* base_ptr = img_buffer + y * img_pitch;
    double gaussian_pixel = 0.0;
    for (int i = -mid_kernel; i <= mid_kernel; i++) {
        for (int j = -mid_kernel; j <= mid_kernel; j++) {
            if (i + x < 0 or i + x >= width or j + y < 0 or j + y >= height)
                continue;
            std::uint8_t cell = ((rgba8_t*)(base_ptr + j * img_pitch))[i + x].r;
            gaussian_pixel += gaussian_kernel[kernel_size * (j + mid_kernel) + (i + mid_kernel)] * cell;
        }
    }
    std::uint8_t cast_gaussian_pixel = (std::uint8_t) gaussian_pixel;

    // apply pixel value
    rgba8_t* base_ptr2 = (rgba8_t*)(res_buffer + y * res_pitch);
    base_ptr2[x] = rgba8_t{cast_gaussian_pixel, cast_gaussian_pixel, cast_gaussian_pixel, 255};
}

// Device code
__global__ void gpu_gray_scale(char* buffer, int width, int height, size_t pitch)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    // get cell
    rgba8_t*  lineptr = (rgba8_t*)(buffer + y * pitch);
    rgba8_t cell = lineptr[x];

    // get gray scale
    std::uint8_t gray = static_cast<std::uint8_t>(0.3 * cell.r + 0.59 * cell.g + 0.11 * cell.b);

    // assign gray pixel
    lineptr[x] = {gray, gray, gray, 255};
}

std::vector<std::vector<int>> render(char* ref_buffer, int width, int height, std::ptrdiff_t stride, char* img_buffer)
{
    hipError_t rc = hipSuccess;

    // Allocate device memory and copy reference image to device memory
    char*  devRefBuffer;
    size_t pitchRef;

    rc = hipMallocPitch(&devRefBuffer, &pitchRef, width * sizeof(rgba8_t), height);
    if (rc)
        abortError("Fail buffer allocation");

    rc = hipMemcpy2D(devRefBuffer, pitchRef, ref_buffer, stride, width * sizeof(rgba8_t), height, hipMemcpyHostToDevice);
    if (rc)
        abortError("Fail buffer allocation");

    // Allocate device memory and copy current image to device memory
    char*  devImgBuffer;
    size_t pitchImg;

    rc = hipMallocPitch(&devImgBuffer, &pitchImg, width * sizeof(rgba8_t), height);
    if (rc)
        abortError("Fail buffer allocation");

    rc = hipMemcpy2D(devImgBuffer, pitchImg, img_buffer, stride, width * sizeof(rgba8_t), height, hipMemcpyHostToDevice);
    if (rc)
        abortError("Fail buffer allocation");

    // Allocate device memory and create/copy gaussian kernel
    double*  gaussianKernel;
    int kernel_size = 5; // gaussian kernel of size 5

    rc = hipMalloc(&gaussianKernel, kernel_size * sizeof(double) * kernel_size);
    if (rc)
        abortError("Fail buffer allocation");

    // get the gaussian kernel
    double sigma = 1.0;
    double* kernel = (double*)malloc(sizeof(double) * kernel_size * kernel_size);
    gaussian_kernel(kernel, sigma, kernel_size);

    rc = hipMemcpy(gaussianKernel, kernel, kernel_size * sizeof(double) * kernel_size, hipMemcpyHostToDevice);
    if (rc)
        abortError("Fail buffer allocation");

    free(kernel);

    // Allocate device memory to store tmp images
    char* devTmpBuffer;
    char* tmp_buff;
    size_t pitchTmp, tmp_pitch;

    rc = hipMallocPitch(&devTmpBuffer, &pitchTmp, width * sizeof(rgba8_t), height);
    if (rc)
        abortError("Fail buffer allocation");

    // Run the kernel with blocks of size 32 x 32
    {
        int bsize = 32;
        int w     = std::ceil((float)width / bsize);
        int h     = std::ceil((float)height / bsize);

        spdlog::debug("running kernel of size ({},{})", w, h);

        dim3 dimBlock(bsize, bsize);
        dim3 dimGrid(w, h);

        // apply gray scale to images
        gpu_gray_scale<<<dimGrid, dimBlock>>>(devRefBuffer, width, height, pitchRef);
        if (hipPeekAtLastError())
            abortError("Computation Error");
        gpu_gray_scale<<<dimGrid, dimBlock>>>(devImgBuffer, width, height, pitchImg);
        if (hipPeekAtLastError())
            abortError("Computation Error");

        // apply gaussian blur to images
        gpu_gaussian_blur<<<dimGrid, dimBlock>>>(devRefBuffer, width, height, pitchRef, gaussianKernel, kernel_size, devTmpBuffer, pitchTmp);
        if (hipPeekAtLastError())
            abortError("Computation Error");
        // Swap buffers
        tmp_buff = devRefBuffer;
        tmp_pitch = pitchRef;
        devRefBuffer = devTmpBuffer;
        pitchRef = pitchTmp;
        devTmpBuffer = tmp_buff;
        pitchTmp = tmp_pitch;

        gpu_gaussian_blur<<<dimGrid, dimBlock>>>(devImgBuffer, width, height, pitchImg, gaussianKernel, kernel_size, devTmpBuffer, pitchTmp);
        if (hipPeekAtLastError())
            abortError("Computation Error");

        // Swap buffers
        tmp_buff = devImgBuffer;
        tmp_pitch = pitchImg;
        devImgBuffer = devTmpBuffer;
        pitchImg = pitchTmp;
        devTmpBuffer = tmp_buff;
        pitchTmp = tmp_pitch;
    }

    // Copy back to main memory
    rc = hipMemcpy2D(img_buffer, stride, devImgBuffer, pitchImg, width * sizeof(rgba8_t), height, hipMemcpyDeviceToHost);
    if (rc)
        abortError("Unable to copy buffer back to memory");

    // Free
    rc = hipFree(devRefBuffer);
    if (rc)
        abortError("Unable to free memory");

    // Free
    rc = hipFree(devImgBuffer);
    if (rc)
        abortError("Unable to free memory");

    // Free
    rc = hipFree(gaussianKernel);
    if (rc)
        abortError("Unable to free memory");

    return {};
}
