#include "hip/hip_runtime.h"
#include "render.hpp"
#include <spdlog/spdlog.h>
#include <cassert>

    [[gnu::noinline]]
void _abortError(const char* msg, const char* fname, int line)
{
    hipError_t err = hipGetLastError();
    spdlog::error("{} ({}, line: {})", msg, fname, line);
    spdlog::error("Error {}: {}", hipGetErrorName(err), hipGetErrorString(err));
    std::exit(1);
}

#define abortError(msg) _abortError(msg, __FUNCTION__, __LINE__)


// Device code
__global__ void gpu_gray_scale(char* buffer, int width, int height, size_t pitch)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    // get cell
    rgba8_t*  lineptr = (rgba8_t*)(buffer + y * pitch);
    rgba8_t cell = lineptr[x];

    // get gray scale
    std::uint8_t gray = static_cast<std::uint8_t>(0.3 * cell.r + 0.59 * cell.g + 0.11 * cell.b);

    // assign gray pixel
    lineptr[x] = {gray, gray, gray, 255};
}

std::vector<std::vector<int>> render(char* ref_buffer, int width, int height, std::ptrdiff_t stride, char* img_buffer)
{
    hipError_t rc = hipSuccess;

    // Allocate device memory and copy reference image to device memory
    char*  devRefBuffer;
    size_t pitchRef;

    rc = hipMallocPitch(&devRefBuffer, &pitchRef, width * sizeof(rgba8_t), height);
    if (rc)
        abortError("Fail buffer allocation");

    rc = hipMemcpy2D(devRefBuffer, pitchRef, ref_buffer, stride, width * sizeof(rgba8_t), height, hipMemcpyHostToDevice);
    if (rc)
        abortError("Fail buffer allocation");

    // Allocate device memory and copy current image to device memory
    char*  devImgBuffer;
    size_t pitchImg;

    rc = hipMallocPitch(&devImgBuffer, &pitchImg, width * sizeof(rgba8_t), height);
    if (rc)
        abortError("Fail buffer allocation");

    rc = hipMemcpy2D(devImgBuffer, pitchImg, img_buffer, stride, width * sizeof(rgba8_t), height, hipMemcpyHostToDevice);
    if (rc)
        abortError("Fail buffer allocation");

    // Run the kernel with blocks of size 32 x 32
    {
        int bsize = 32;
        int w     = std::ceil((float)width / bsize);
        int h     = std::ceil((float)height / bsize);

        spdlog::debug("running kernel of size ({},{})", w, h);

        dim3 dimBlock(bsize, bsize);
        dim3 dimGrid(w, h);

       gpu_gray_scale<<<dimGrid, dimBlock>>>(devRefBuffer, width, height, pitchRef);
        if (hipPeekAtLastError())
            abortError("Computation Error");

       gpu_gray_scale<<<dimGrid, dimBlock>>>(devImgBuffer, width, height, pitchImg);
        if (hipPeekAtLastError())
            abortError("Computation Error");
    }

    // Copy back to main memory
    rc = hipMemcpy2D(img_buffer, stride, devImgBuffer, pitchImg, width * sizeof(rgba8_t), height, hipMemcpyDeviceToHost);
    if (rc)
        abortError("Unable to copy buffer back to memory");

    // Free
    rc = hipFree(devRefBuffer);
    if (rc)
        abortError("Unable to free memory");

    // Free
    rc = hipFree(devImgBuffer);
    if (rc)
        abortError("Unable to free memory");

    return {};
}
