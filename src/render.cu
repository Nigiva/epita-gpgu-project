#include "hip/hip_runtime.h"
#include "render.hpp"
#include <spdlog/spdlog.h>
#include <cassert>
#include "utils.hpp"
#include <math.h>
#include <thrust/device_vector.h>
#include <iostream>

#define ABS_MIN(a, b) ((a>=b)?(a-b):(b-a))

    [[gnu::noinline]]
void _abortError(const char* msg, const char* fname, int line)
{
    hipError_t err = hipGetLastError();
    spdlog::error("{} ({}, line: {})", msg, fname, line);
    spdlog::error("Error {}: {}", hipGetErrorName(err), hipGetErrorString(err));
    std::exit(1);
}

#define abortError(msg) _abortError(msg, __FUNCTION__, __LINE__)

// Device code
__global__ void gpu_gaussian_blur(char* img_buffer, int width, int height, size_t img_pitch, double* gaussian_kernel, int kernel_size, char* res_buffer, size_t res_pitch)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    int mid_kernel = (kernel_size - 1) / 2;

    // get pixel value
    char* base_ptr = img_buffer + y * img_pitch;
    double gaussian_pixel = 0.0;
    for (int i = -mid_kernel; i <= mid_kernel; i++) {
        for (int j = -mid_kernel; j <= mid_kernel; j++) {
            if (i + x < 0 or i + x >= width or j + y < 0 or j + y >= height)
                continue;
            std::uint8_t cell = ((rgba8_t*)(base_ptr + j * img_pitch))[i + x].r;
            gaussian_pixel += gaussian_kernel[kernel_size * (j + mid_kernel) + (i + mid_kernel)] * cell;
        }
    }
    std::uint8_t cast_gaussian_pixel = (std::uint8_t) gaussian_pixel;

    // apply pixel value
    rgba8_t* base_ptr2 = (rgba8_t*)(res_buffer + y * res_pitch);
    base_ptr2[x] = rgba8_t{cast_gaussian_pixel, cast_gaussian_pixel, cast_gaussian_pixel, 255};
}

// Device code
__global__ void gpu_gray_scale(char* buffer, int width, int height, size_t pitch)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    // get cell
    rgba8_t*  lineptr = (rgba8_t*)(buffer + y * pitch);
    rgba8_t cell = lineptr[x];

    // get gray scale
    std::uint8_t gray = static_cast<std::uint8_t>(0.3 * cell.r + 0.59 * cell.g + 0.11 * cell.b);

    // assign gray pixel
    lineptr[x] = {gray, gray, gray, 255};
}


__global__ void gpu_difference(char* ref_buffer, int width, int height, size_t ref_pitch, char* img_buffer, size_t img_pitch)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    // get reference and image cell
    rgba8_t* ref_lineptr = (rgba8_t*)(ref_buffer + y * ref_pitch);
    rgba8_t ref_cell = ref_lineptr[x];
    rgba8_t* img_lineptr = (rgba8_t*)(img_buffer + y * img_pitch);
    rgba8_t img_cell = img_lineptr[x];

    std::uint8_t r = ABS_MIN(ref_cell.r, img_cell.r);
    std::uint8_t g = ABS_MIN(ref_cell.g, img_cell.g);
    std::uint8_t b = ABS_MIN(ref_cell.b, img_cell.b);

    // assign diff pixel to image buffer
    img_lineptr[x] = {r, g, b, 255};
}

__global__ void erosion_dilation(char *img_buffer, int width, int height, int img_pitch, int radius, bool is_square, bool is_erosion, char* res_buffer, int res_pitch, bool is_baseline)
{
    if (!is_baseline){
        // Define shared memory

        extern __shared__ std::uint8_t tile[];


        int x = blockDim.x * blockIdx.x + threadIdx.x;
        int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x >= width || y >= height)
            return;

        int base_x = blockDim.x * blockIdx.x;
        int base_y = blockDim.y * blockIdx.y;

        std::uint8_t* block_ptr = (std::uint8_t*)((img_buffer + (base_x - radius) * sizeof(rgba8_t)) + (base_y - radius) * img_pitch);
        int base_width = blockDim.x + 2 * radius;

        for (int j = threadIdx.y;  j < base_width; j+= blockDim.y){
            for (int i = threadIdx.x;  i < base_width; i+= blockDim.x){
                if (base_x - radius + i >= 0 and base_y - radius + j >= 0 and base_x - radius + i < width and base_y - radius + j < height)
                    tile[i + j * base_width] = block_ptr[i * sizeof(rgba8_t) + j * img_pitch];
            }
        }
        __syncthreads();

        std::uint8_t val = (is_erosion) ? 255 : 0;

        // iteration on each pixel around current pixel
        for (int i = -radius; i <= radius; i++){
            for (int j = -radius; j <= radius; j++){
                // if outside the image
                if (i + x < 0 or i + x >= width or j + y < 0 or j + y >= height)
                    continue;

                // if disc and not in it
                if (!is_square and !((int)sqrtf(i*i + j*j) <= radius))
                    continue;

                std::uint8_t cell = tile[x - base_x + i + radius + (y - base_y + j + radius) * base_width];
                if (is_erosion and val > cell){
                    val = cell;
                }
                else if (!is_erosion and val < cell){
                    val = cell;
                }
            }
        }

        // apply pixel value
        rgba8_t* base_ptr2 = (rgba8_t*)(res_buffer + y * res_pitch);
        base_ptr2[x] = rgba8_t{val, val, val, 255};
    }
    else{
        int x = blockDim.x * blockIdx.x + threadIdx.x;
        int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x >= width || y >= height)
            return;

        char* base_ptr = img_buffer + y * img_pitch;

        std::uint8_t val = (is_erosion) ? 255 : 0;

        // iteration on each pixel around current pixel
        for (int i = -radius; i <= radius; i++){
            for (int j = -radius; j <= radius; j++){
                // if outside the image
                if (i + x < 0 or i + x >= width or j + y < 0 or j + y >= height)
                    continue;

                // if disc and not in it
                if (!is_square and !((int)sqrtf(i*i + j*j) <= radius))
                    continue;
                std::uint8_t cell = ((rgba8_t*)(base_ptr + j * img_pitch))[i + x].r;
                if (is_erosion and val > cell){
                    val = cell;
                }
                else if (!is_erosion and val < cell){
                    val = cell;
                }
            }
        }

        // apply pixel value
        rgba8_t* base_ptr2 = (rgba8_t*)(res_buffer + y * res_pitch);
        base_ptr2[x] = rgba8_t{val, val, val, 255};
    }
}

__global__ void histogram(char* img_buffer, int width, int height, int img_pitch, int* histo)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    std::uint8_t cellValue = ((rgba8_t*)(img_buffer + y * img_pitch))[x].r;
    atomicAdd(histo + cellValue, 1);
}

__global__ void thresholding(char* img_buffer, int width, int height, int img_pitch, int threshold, int* L)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= width || y >= height)
        return;


    rgba8_t* lineptr = (rgba8_t*)(img_buffer + y * img_pitch);
    if (lineptr[x].r < threshold){
        L[y * width + x] = 0;
    }
    else {
        L[y * width + x] = y * width + x + 1;
    }
}

__global__ void propagate_relabeling(int* L, int width, int height, bool* is_changed, bool relabeling, int* nb_components){

    __shared__ bool changed;
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    if (L[y * width + x] == 0)
        return;

    if (relabeling and L[y * width + x] == y * width + x + 1){
        *is_changed = true;
        // relabeling
        L[y * width + x] = atomicAdd(nb_components, 1) + 1;
        return;
    }

    int mid_kernel = 1;

    // Propagate
    do
    {
        changed = false;
        __syncthreads();
        // look pixels around
        for (int i = -mid_kernel; i <= mid_kernel; i++) {
            for (int j = -mid_kernel; j <= mid_kernel; j++) {
                if (i + x < 0 or i + x >= width or j + y < 0 or j + y >= height)
                    continue;
                if (L[(j+y) * width + i+x] == 0)
                    continue;
                if (L[(j+y) * width + i+x] < L[y * width + x]){
                    L[y * width + x] = L[(j+y) * width + i+x];
                    *is_changed = true;
                    changed = true;
                }
            }
        }
        __syncthreads();
    }while (changed);
}
__global__ void get_bbox(int* L, int width, int height, int* max_values, int* bbox, char* img_buffer, int pitch)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= width || y >= height)
        return;
    if (L[y * width + x] == 0)
        return;

    int component = L[y * width + x];

    uint8_t value = ((rgba8_t*)(img_buffer + y * pitch))[x].r;

    // put max value for the current component
    atomicMax(max_values + component - 1, (int)value);

    // change bbox values if necessary

    if (y - 1 >= 0 and value != ((rgba8_t*)(img_buffer + (y - 1) * pitch))[x].r)
        atomicMin(bbox + (component - 1) * 4 + 1, y);
    if (y + 1 < height and value != ((rgba8_t*)(img_buffer + (y + 1) * pitch))[x].r)
        atomicMax(bbox + (component - 1) * 4 + 3, y);
    if (x - 1 >= 0 and value != ((rgba8_t*)(img_buffer + y * pitch))[x - 1].r)
        atomicMin(bbox + (component - 1) * 4, x);
    if (x + 1 < width and value != ((rgba8_t*)(img_buffer + y * pitch))[x + 1].r)
        atomicMax(bbox + (component - 1) * 4 + 2, x);
}

std::vector<std::vector<int>> render(char* ref_buffer, int width, int height, std::ptrdiff_t stride, char* img_buffer, bool is_baseline)
{
    hipError_t rc = hipSuccess;

    // Allocate device memory and copy reference image to device memory
    char*  devRefBuffer;
    size_t pitchRef;

    rc = hipMallocPitch(&devRefBuffer, &pitchRef, width * sizeof(rgba8_t), height);
    if (rc)
        abortError("Fail buffer allocation");

    rc = hipMemcpy2D(devRefBuffer, pitchRef, ref_buffer, stride, width * sizeof(rgba8_t), height, hipMemcpyHostToDevice);
    if (rc)
        abortError("Fail buffer allocation");

    // Allocate device memory and copy current image to device memory
    char*  devImgBuffer;
    size_t pitchImg;

    rc = hipMallocPitch(&devImgBuffer, &pitchImg, width * sizeof(rgba8_t), height);
    if (rc)
        abortError("Fail buffer allocation");

    rc = hipMemcpy2D(devImgBuffer, pitchImg, img_buffer, stride, width * sizeof(rgba8_t), height, hipMemcpyHostToDevice);
    if (rc)
        abortError("Fail buffer allocation");

    // Allocate device memory and create/copy gaussian kernel
    double*  gaussianKernel;
    int kernel_size = 5; // gaussian kernel of size 5

    rc = hipMalloc(&gaussianKernel, kernel_size * sizeof(double) * kernel_size);
    if (rc)
        abortError("Fail buffer allocation");

    // get the gaussian kernel
    double sigma = 1.0;
    double* kernel = (double*)malloc(sizeof(double) * kernel_size * kernel_size);
    gaussian_kernel(kernel, sigma, kernel_size);

    rc = hipMemcpy(gaussianKernel, kernel, kernel_size * sizeof(double) * kernel_size, hipMemcpyHostToDevice);
    if (rc)
        abortError("Fail buffer allocation");

    free(kernel);

    // Allocate device memory to store tmp images
    char* devTmpBuffer;
    char* tmp_buff;
    size_t pitchTmp, tmp_pitch;

    rc = hipMallocPitch(&devTmpBuffer, &pitchTmp, width * sizeof(rgba8_t), height);
    if (rc)
        abortError("Fail buffer allocation");

    // Allocate device memory for histogram
    int* histoBuffer;

    rc = hipMalloc(&histoBuffer, 256 * sizeof(int));
    if (rc)
        abortError("Fail buffer allocation");
    rc = hipMemset(histoBuffer, 0, 256 * sizeof(int));
    if (rc)
        abortError("Fail buffer allocation");

    // Allocate device memory for L
    int* L;

    rc = hipMalloc(&L, width * height * sizeof(int));
    if (rc)
        abortError("Fail buffer allocation");

    // Allocate device memory for share information between CPU and GPU
    bool* is_changed;

    rc = hipMalloc(&is_changed, sizeof(bool));
    if (rc)
        abortError("Fail buffer allocation");
    rc = hipMemset(is_changed, true, sizeof(bool));
    if (rc)
        abortError("Fail buffer allocation");

    int* nb_components;

    rc = hipMalloc(&nb_components, sizeof(int));
    if (rc)
        abortError("Fail buffer allocation");
    rc = hipMemset(nb_components, 0, sizeof(int));
    if (rc)
        abortError("Fail buffer allocation");

    std::vector<std::vector<int>> result;


    // Run the kernel with blocks of size 32 x 32
    {
        int bsize = 32;
        int w     = std::ceil((float)width / bsize);
        int h     = std::ceil((float)height / bsize);

        spdlog::debug("running kernel of size ({},{})", w, h);

        dim3 dimBlock(bsize, bsize);
        dim3 dimGrid(w, h);

        // apply gray scale to images
        gpu_gray_scale<<<dimGrid, dimBlock>>>(devRefBuffer, width, height, pitchRef);
        if (hipPeekAtLastError())
            abortError("Computation Error");
        gpu_gray_scale<<<dimGrid, dimBlock>>>(devImgBuffer, width, height, pitchImg);
        if (hipPeekAtLastError())
            abortError("Computation Error");

        // apply gaussian blur to images
        gpu_gaussian_blur<<<dimGrid, dimBlock>>>(devRefBuffer, width, height, pitchRef, gaussianKernel, kernel_size, devTmpBuffer, pitchTmp);
        if (hipPeekAtLastError())
            abortError("Computation Error");
        // Swap buffers
        tmp_buff = devRefBuffer;
        tmp_pitch = pitchRef;
        devRefBuffer = devTmpBuffer;
        pitchRef = pitchTmp;
        devTmpBuffer = tmp_buff;
        pitchTmp = tmp_pitch;

        gpu_gaussian_blur<<<dimGrid, dimBlock>>>(devImgBuffer, width, height, pitchImg, gaussianKernel, kernel_size, devTmpBuffer, pitchTmp);
        if (hipPeekAtLastError())
            abortError("Computation Error");

        // Swap buffers
        tmp_buff = devImgBuffer;
        tmp_pitch = pitchImg;
        devImgBuffer = devTmpBuffer;
        pitchImg = pitchTmp;
        devTmpBuffer = tmp_buff;
        pitchTmp = tmp_pitch;

        // difference
        gpu_difference<<<dimGrid, dimBlock>>>(devRefBuffer, width, height, pitchRef, devImgBuffer, pitchImg);

        // calculate adaptative closing opening radius
        double closing_radius = width * height * 10 / (1920 * 1080);
        double opening_radius = width * height * 25 / (1920 * 1080);

        // Run optimized version or not
        if (!is_baseline){
            // perform morphology closing and opening
            // closing
            erosion_dilation<<<dimGrid, dimBlock, (bsize + 2 * (int)closing_radius) * (bsize + 2 * (int)closing_radius) * sizeof(std::uint8_t)>>>(devImgBuffer, width, height, pitchImg, (int)closing_radius, false, false, devTmpBuffer, pitchTmp, false);
            erosion_dilation<<<dimGrid, dimBlock, (bsize + 2 * (int)closing_radius) * (bsize + 2 * (int)closing_radius) * sizeof(std::uint8_t)>>>(devTmpBuffer, width, height, pitchTmp, (int)closing_radius, false, true, devImgBuffer, pitchImg, false);
            //opening
            erosion_dilation<<<dimGrid, dimBlock, (bsize + 2 * (int)opening_radius) * (bsize + 2 * (int)opening_radius) * sizeof(std::uint8_t)>>>(devImgBuffer, width, height, pitchImg, (int)opening_radius, false, true, devTmpBuffer, pitchTmp, false);
            erosion_dilation<<<dimGrid, dimBlock, (bsize + 2 * (int)opening_radius) * (bsize + 2 * (int)opening_radius) * sizeof(std::uint8_t)>>>(devTmpBuffer, width, height, pitchTmp, (int)opening_radius, false, false, devImgBuffer, pitchImg, false);
        }
        else{
            // closing
            erosion_dilation<<<dimGrid, dimBlock>>>(devImgBuffer, width, height, pitchImg, (int)closing_radius, false, false, devTmpBuffer, pitchTmp, true);
            erosion_dilation<<<dimGrid, dimBlock>>>(devTmpBuffer, width, height, pitchTmp, (int)closing_radius, false, true, devImgBuffer, pitchImg, true);
            //opening
            erosion_dilation<<<dimGrid, dimBlock>>>(devImgBuffer, width, height, pitchImg, (int)opening_radius, false, true, devTmpBuffer, pitchTmp, true);
            erosion_dilation<<<dimGrid, dimBlock>>>(devTmpBuffer, width, height, pitchTmp, (int)opening_radius, false, false, devImgBuffer, pitchImg, true);

        }
        // get histogram of the image
        histogram<<<dimGrid, dimBlock>>>(devImgBuffer, width, height, pitchImg, histoBuffer);

        // copy histogram from device to host
        int* histoHostBuffer = (int*)malloc(256 * sizeof(int));
        // Here is the probleme
        rc = hipMemcpy(histoHostBuffer, histoBuffer, 256 * sizeof(int), hipMemcpyDeviceToHost);

        if (rc)
            abortError("Unable to copy buffer back to memory");

        // otsu: first threshold
        // (cumpute on Host !)
        int threshold_1 = otsu(width, height, histoHostBuffer);

        if (threshold_1 < 5)
            threshold_1 = 5;

        // puts zeros in the histogram for elements in [0:threshold_1]
        int N = 0;
        for (int i = 0; i <= threshold_1; i++){
            N += histoHostBuffer[i];
            histoHostBuffer[i] = 0;
        }

        // otsu: second threshold
        // (cumpute on Host !)
        int threshold_2 = otsu(1, width * height - N, histoHostBuffer);

        if (threshold_2 < 10)
            threshold_2 = 10;

        free(histoHostBuffer);


        // apply thresholding
        thresholding<<<dimGrid, dimBlock>>>(devImgBuffer, width, height, pitchImg, threshold_1, L);

        // Apply propagate
        bool* is_changed_host = (bool*)malloc(sizeof(bool));
        *is_changed_host = true;

        bool create_comp = false;
        for (int i = 0; i <= 1; i++){
            create_comp = (bool) i;
            while (*is_changed_host){
                rc = hipMemset(is_changed, false, sizeof(bool));
                if (rc)
                    abortError("Fail buffer allocation");
                propagate_relabeling<<<dimGrid, dimBlock>>>(L, width, height, is_changed, create_comp, nb_components);
                create_comp = false;

                rc = hipMemcpy(is_changed_host, is_changed, sizeof(bool), hipMemcpyDeviceToHost);
                if (rc)
                    abortError("Unable to copy buffer back to memory");
            }
            *is_changed_host = true;
        }
        free(is_changed_host);


        int* nb_components_host = (int*)malloc(sizeof(int));
        rc = hipMemcpy(nb_components_host, nb_components, sizeof(int), hipMemcpyDeviceToHost);
        if (rc)
            abortError("Unable to copy buffer back to memory");

        if (*nb_components_host != 0){
            // Apply bbox
            int* bbox;
            rc = hipMalloc(&bbox, 4 * (*nb_components_host) * sizeof(int));
            if (rc)
                abortError("Fail buffer allocation");
            rc = hipMemset(bbox, 0, 4 * (*nb_components_host) * sizeof(int));
            if (rc)
                abortError("Fail buffer allocation");

            for (int i = 0; i < *nb_components_host; i++)
            {
                rc = hipMemset(bbox + i * 4, 127, 2 * sizeof(int));
                if (rc)
                    abortError("Fail buffer allocation");
            }

            int* max_values;
            rc = hipMalloc(&max_values, (*nb_components_host) * sizeof(int));
            if (rc)
                abortError("Fail buffer allocation");
            rc = hipMemset(max_values, 0, (*nb_components_host) * sizeof(int));
            if (rc)
                abortError("Fail buffer allocation");

            get_bbox<<<dimGrid, dimBlock>>>(L, width, height, max_values, bbox, devImgBuffer, pitchImg);
            if (hipPeekAtLastError())
                abortError("Computation Error");

            int* bbox_host = (int*)malloc(4 * (*nb_components_host) * sizeof(int));
            int* max_values_host = (int*)malloc((*nb_components_host) * sizeof(int));

            rc = hipMemcpy(bbox_host, bbox, 4 * (*nb_components_host) * sizeof(int), hipMemcpyDeviceToHost);
            if (rc)
                abortError("Unable to copy buffer back to memory");

            rc = hipMemcpy(max_values_host, max_values, (*nb_components_host) * sizeof(int), hipMemcpyDeviceToHost);
            if (rc)
                abortError("Unable to copy buffer back to memory");

            for (int i = 0; i < *nb_components_host; i++)
            {
                if (max_values_host[i] >= threshold_2)
                {
                    std::vector<int> cur_bbox = {bbox_host[i * 4], bbox_host[i * 4 + 1], bbox_host[i * 4 + 2] - bbox_host[i * 4], bbox_host[i * 4 + 3] - bbox_host[i * 4 + 1]};
                    result.push_back(cur_bbox);
                }
            }

            // Free
            rc = hipFree(bbox);
            if (rc)
                abortError("Unable to free memory");
            rc = hipFree(max_values);
            if (rc)
                abortError("Unable to free memory");

            free(bbox_host);
            free(max_values_host);
        }

        free(nb_components_host);
    }
    // Copy back to main memory
    rc = hipMemcpy2D(img_buffer, stride, devImgBuffer, pitchImg, width * sizeof(rgba8_t), height, hipMemcpyDeviceToHost);
    if (rc)
        abortError("Unable to copy buffer back to memory");

    // Free
    rc = hipFree(devRefBuffer);
    if (rc)
        abortError("Unable to free memory");

    // Free
    rc = hipFree(devImgBuffer);
    if (rc)
        abortError("Unable to free memory");

    // Free
    rc = hipFree(gaussianKernel);
    if (rc)
        abortError("Unable to free memory");

    // Free
    rc = hipFree(devTmpBuffer);
    if (rc)
        abortError("Unable to free memory");

    // Free
    rc = hipFree(histoBuffer);
    if (rc)
        abortError("Unable to free memory");

    // Free
    rc = hipFree(L);
    if (rc)
        abortError("Unable to free memory");
    // Free
    rc = hipFree(is_changed);
    if (rc)
        abortError("Unable to free memory");

    rc = hipFree(nb_components);
    if (rc)
        abortError("Unable to free memory");
    return result;
}
